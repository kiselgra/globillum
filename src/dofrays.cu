#include "hip/hip_runtime.h"
#include "dofrays.h"

#include <librta/basic_types.h>
#include <librta/cuda-kernels.h>
#include <librta/cuda-vec.h>

namespace rta {
	namespace cuda {
		namespace k {
			__global__ void setup_shirley_lens_rays(float *dirs, float *orgs, float *maxts, 
													float fovy, float aspect, int w, int h, float3 view_dir, float3 pos, float3 up, float maxt,
													float focus_distance, float aperture, float eye_to_lens, gi::cuda::mt_pool3f uniform_random_01) {
				int2 gid = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
				if (gid.x >= w || gid.y >= h)
					return;
				int id = gid.y*w+gid.x;

				maxts[gid.y * w + gid.x] = maxt;
				fovy /= 2.0;
				float height = tanf(M_PI * fovy / 180.0f);
				float width = aspect * height;
				
				float u_s = (((float)gid.x+0.5f)/(float)w) * 2.0f - 1.0f;	// \in (-1,1)
				float v_s = (((float)gid.y+0.5f)/(float)h) * 2.0f - 1.0f;
				u_s = width * u_s;	// \in (-pw/2, pw/2)
				v_s = height * v_s;
			
				float3 vd = view_dir;
				float3 vu = up;
				float3 W, TxW, U, V;
				div_vec3f_by_scalar(&W, &vd, length_of_vec3f(&vd));
				cross_vec3f(&TxW, &vu, &W);
				div_vec3f_by_scalar(&U, &TxW, length_of_vec3f(&TxW));
				cross_vec3f(&V, &W, &U);

				float3 dir = make_float3(0,0,0), tmp;
				mul_vec3f_by_scalar(&dir, &U, u_s);
				mul_vec3f_by_scalar(&tmp, &V, v_s);
				add_components_vec3f(&dir, &dir, &tmp);
				add_components_vec3f(&dir, &dir, &W);
				normalize_vec3f(&dir);
				
				float3 pos_on_focal_plane = pos + dir*(1.0f/(dir|view_dir))*focus_distance;
				
				if (gid.x == 200 && gid.y == 100) {
					printf("dir %6.6f %6.6f %6.6f %f\n", dir.x, dir.y, dir.z, (dir|view_dir));
					printf("pos %6.6f %6.6f %6.6f\n", pos.x, pos.y, pos.z);
					printf("pos %6.6f %6.6f %6.6f\n", pos_on_focal_plane.x, pos_on_focal_plane.y, pos_on_focal_plane.z);
				}
				gi::cuda::random_sampler_path_info dummy;
				float2 jitter;
				int i;
				do {
					float3 random = next_random3f(uniform_random_01, id+17*i, dummy);
					jitter = make_float2(random.x-0.5f, random.y-0.5f);
				} while (jitter.x*jitter.x + jitter.y*jitter.y > 1.0f);

				float3 jitter_pos = pos + U*jitter.x*aperture + V*jitter.y*aperture;
				dir = (pos_on_focal_plane - jitter_pos);
				normalize_vec3f(&dir);

				dirs[3*(gid.y * w + gid.x)+0] = dir.x;
				dirs[3*(gid.y * w + gid.x)+1] = dir.y;
				dirs[3*(gid.y * w + gid.x)+2] = dir.z;
				orgs[3*(gid.y * w + gid.x)+0] = jitter_pos.x;
				orgs[3*(gid.y * w + gid.x)+1] = jitter_pos.y;
				orgs[3*(gid.y * w + gid.x)+2] = jitter_pos.z;

			}
		}
			
		void setup_shirley_lens_rays(float *dirs, float *orgs, float *maxts, 
									 float fovy, float aspect, int w, int h, float3 *view_dir, float3 *pos, float3 *up, float maxt,
									 float focus_distance, float aperture, float eye_to_lens, gi::cuda::mt_pool3f uniform_random_01) {
				checked_cuda(hipPeekAtLastError());
				dim3 threads(16, 16);
				dim3 blocks = block_configuration_2d(w, h, threads);
				k::setup_shirley_lens_rays<<<blocks, threads>>>(dirs, orgs, maxts, fovy, aspect, w, h, *view_dir, *pos, *up, maxt,
																focus_distance, aperture, eye_to_lens, uniform_random_01);
				checked_cuda(hipPeekAtLastError());
				checked_cuda(hipDeviceSynchronize());
		}
	
	}
}
